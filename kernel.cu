#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <fstream>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
hipError_t decodeWithCuda(const char*src, char*dst, const unsigned int src_size, const unsigned int dst_size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}


__global__ void decodeKernel(const char* src, char* dst,int sum)
{
    int i = threadIdx.x;
    
}


int main()
{
    std::string file("./base64.data");
    std::ifstream input(file, std::ios::binary|std::ios::ate);
    unsigned size = input.tellg();
    input.seekg(std::ios::beg);

    char* base64 = new char[size];
    input.read(base64, size);
    input.close();


    int goups = size / 3 + 1;

    char* host_dst = nullptr;
    hipError_t cudaStatus = hipHostAlloc((void**)&host_dst, goups * 4 * sizeof(int), hipHostMallocDefault);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }




    cudaStatus = decodeWithCuda(base64, host_dst, size, goups);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }


    std::ofstream output("output.jpeg", std::ios::binary);
    output.write(host_dst, goups);
    output.close();


    free(base64);
    hipHostFree(host_dst);

   

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}



hipError_t decodeWithCuda(const char*& src, char*& dst, const unsigned int src_size,const unsigned int dst_size)
{
    int* dev_src = 0;
    int* dev_dst = 0;
    hipError_t cudaStatus;

   


    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_src, src_size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }




    cudaStatus = hipMalloc((void**)&dev_dst, goups * 4 * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }





    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_src, src, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }



    // Launch a kernel on the GPU with one thread for each element.
    decodeKernel << <1, size >> > (dev_src, dev_dst, goups);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
       
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(dev_dst, host_dst, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_src);
    hipFree(dev_dst);
    hipHostFree(host_dst);

    return cudaStatus;
}




// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
